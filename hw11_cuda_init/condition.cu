#include "hip/hip_runtime.h"

#include <stdio.h>

__device__ int res = 0; 

__device__ bool condition(int i, int j) {
    return i == j;
}

__global__ void countThreads(int rows, int cols) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < rows && j < cols) {
        if (condition(i, j)) {
            atomicAdd(&res, 1);
        }
    }
}

int main() {
    int rows = 10;
    int cols = 10;

    int numThreadsPerBlock = 16;
    int numBlocksX = (cols + numThreadsPerBlock - 1) / numThreadsPerBlock;
    int numBlocksY = (rows + numThreadsPerBlock - 1) / numThreadsPerBlock;

    dim3 gridSize(numBlocksX, numBlocksY);
    dim3 blockSize(numThreadsPerBlock, numThreadsPerBlock);

    countThreads<<<gridSize, blockSize>>>(rows, cols);

    hipDeviceSynchronize();

    int result;
    hipMemcpyFromSymbol(&result, HIP_SYMBOL(res), sizeof(int));

    printf("Number of threads satisfying the condition: %d\n", result);

    getchar();

    return 0;
}